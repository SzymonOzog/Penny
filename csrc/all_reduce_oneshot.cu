#include "hip/hip_runtime.h"
#include "device_host_transport/nvshmem_common_transport.h"
#include "host/nvshmem_api.h"
#include "host/nvshmemx_api.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <nvshmem.h>
#include <nvshmemx.h>
#include "common.h"

template <typename scalar_t, int N_PES = 8>
__global__ void all_reduce_oneshot_kernel(scalar_t* __restrict__ destination, scalar_t* __restrict__ buffer, scalar_t* __restrict__ output, uint64_t* __restrict__ signal,
        const int packet_size, const int gpus_per_node, int stage)
{
    using P = array_t<scalar_t, 16/sizeof(scalar_t)>;

    const uint32_t block_size = blockDim.x * packet_size;
    const uint32_t pe_off = block_size/sizeof(scalar_t);
    const uint32_t off = blockIdx.z * pe_off;

    const int pe = nvshmem_my_pe();
    const int n_pes = nvshmem_n_pes();

    if (blockIdx.x != pe && blockIdx.y == 0)
    {
            nvshmemx_putmem_signal_nbi_block(destination + pe*pe_off + off*N_PES,
                    buffer + off,
                    block_size, signal+pe + blockIdx.z*N_PES, stage, NVSHMEM_SIGNAL_SET, blockIdx.x);
    }

    for(int tid = 0; tid<N_PES; tid++)
    {
        if (threadIdx.x == tid && tid != pe)
        {
            nvshmem_signal_wait_until(signal+tid + blockIdx.z*N_PES, NVSHMEM_CMP_EQ, stage);
        }
    }

    __syncthreads();
    const uint32_t reduce_size = block_size/(N_PES*gridDim.y);
    const uint32_t reduce_off = (blockIdx.y*gridDim.x + blockIdx.x)*reduce_size/sizeof(scalar_t);
    // if(pe == 0 && blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0)
    //     printf("reducing z %d sz %d, off %d, buff %d, pe_of %d, block_size %d, off %d\n",
    //             blockIdx.z, reduce_size, reduce_off, int(buffer[reduce_off + off]), pe_off, block_size, off);

    for (int i = threadIdx.x; i < reduce_size/(sizeof(P)); i += blockDim.x)
    {
        P res = reinterpret_cast<P*>(buffer + reduce_off + off)[i];
        for (int recv_pe = 0; recv_pe < N_PES; recv_pe++)
        {
            if(recv_pe == pe)
                continue;
            P src = reinterpret_cast<P*>(destination + recv_pe*pe_off + reduce_off + off*N_PES)[i];
            for (int j = 0; j < P::size; j++)
            {
                res.data[j] += float(src.data[j]);
            }
        }
        reinterpret_cast<P*>(output + reduce_off + off)[i] = res;
    }
}

AllReduceOneShot::AllReduceOneShot(half* _buffer, int numel, int packet_size, int block_size, int nnodes, int routes, hipStream_t stream)
    : AllReduce(_buffer, numel, numel*nvshmem_n_pes(), packet_size, block_size, nnodes,
            nvshmem_n_pes() * numel*sizeof(half)/(block_size*packet_size), stream)
{
    grid_dim.x = nvshmem_n_pes();
    grid_dim.y = routes;
    grid_dim.z = (numel*sizeof(half))/(block_size*packet_size);
}
void AllReduceOneShot::run(half* output, hipStream_t stream)
{
    all_reduce_oneshot_kernel<half><<<grid_dim, block_dim, 0, stream>>>(
            destination,
            static_cast<half*>(buffer),
            output,
            signal,
            packet_size,
            gpus_per_node,
            stage
            );
    stage+=2;
}
