#include "hip/hip_runtime.h"
#include "host/nvshmem_api.h"
#include "host/nvshmemx_api.h"
#include <pybind11/functional.h>
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <nvshmem.h>
#include <nvshmemx.h>

template <typename scalar_t>
__global__ void exchange(scalar_t *destination, scalar_t* buffer, uint64_t* signal, int peer, int packet_size) 
{
    const uint64_t off = (blockIdx.x * blockDim.x) * packet_size/sizeof(scalar_t);
    const uint64_t block_size = blockDim.x * packet_size;

    constexpr uint64_t SIG_SYNC = 1;
    nvshmemx_putmem_signal_block(destination + off, buffer + off, block_size, signal + blockIdx.x, SIG_SYNC, NVSHMEM_SIGNAL_SET, peer);
    // nvshmemx_putmem_block(destination + off, buffer + off, block_size, peer);
    // nvshmem_fence();
    // __syncthreads();
    //
    // if (threadIdx.x == 0)
    // {
    //     nvshmemx_signal_op(signal + blockIdx.x, SIG_SYNC, NVSHMEM_SIGNAL_SET, peer);
    // }
    if (threadIdx.x == 0)
        nvshmem_signal_wait_until(signal + blockIdx.x, NVSHMEM_CMP_EQ, SIG_SYNC);
    __syncthreads();

    for (int i = threadIdx.x; i < block_size/(sizeof(float4)); i += blockDim.x)
        reinterpret_cast<float4*>(buffer +off)[i] = reinterpret_cast<float4*>(destination + off)[i];
}

void exchange(torch::Tensor& buffer, int packet_size, int block_size, int peer) 
{
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    half *destination = (half *) nvshmem_malloc(buffer.numel() * sizeof(half));

    nvshmemx_buffer_register(buffer.data_ptr(), buffer.numel() * sizeof(half));
    
    const uint32_t grid_size = std::ceil(buffer.numel()*sizeof(half) / float(packet_size*block_size));

    uint64_t *signal = (uint64_t *) nvshmem_malloc(grid_size * sizeof(uint64_t));
    hipMemset(signal, 0, grid_size*sizeof(uint64_t));

    //sync the memset before running kernel
    nvshmemx_barrier_all_on_stream(stream);

    exchange<<<grid_size, block_size, 0, stream>>>(destination,
            static_cast<half*>(buffer.data_ptr()),
            signal,
            peer,
            packet_size);

    nvshmemx_barrier_all_on_stream(stream);
    // hipMemcpyAsync(buffer.data_ptr(), (void*)destination, buffer.numel() * sizeof(half), hipMemcpyDeviceToDevice, stream);

    hipStreamSynchronize(stream);

    nvshmemx_buffer_unregister(buffer.data_ptr());
    nvshmem_free(destination);
}
