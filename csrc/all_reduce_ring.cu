#include "hip/hip_runtime.h"
#include "device_host_transport/nvshmem_common_transport.h"
#include "host/nvshmem_api.h"
#include "host/nvshmemx_api.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <nvshmem.h>
#include <nvshmemx.h>

template <typename T> __device__ __forceinline__ void swap_cu(T& a, T& b)
{
    T c(a); a=b; b=c;
}

template <typename scalar_t>
__global__ void all_reduce_ring_kernel(scalar_t *destination, scalar_t* buffer, uint64_t* signal, int packet_size, int gpus_per_node) 
{
    const uint64_t base_off = (blockIdx.x * blockDim.x) * packet_size/sizeof(scalar_t);
    const uint64_t block_size = blockDim.x * packet_size;
    const uint64_t chunk_off = (gridDim.x * blockDim.x) * packet_size/sizeof(scalar_t);
    const uint32_t ring_id = blockIdx.y;
    const uint64_t ring_off = ring_id * chunk_off * nvshmem_n_pes();
    const uint64_t off = base_off + ring_off;

    const uint32_t local_rank = nvshmem_my_pe()%gpus_per_node;
    const uint32_t my_node = nvshmem_my_pe()/gpus_per_node;

    int send_peer;
    int recv_peer;
    
    if (local_rank == ring_id)
    {
        send_peer = (nvshmem_my_pe()+gpus_per_node) % nvshmem_n_pes();
        recv_peer = my_node*gpus_per_node + (gpus_per_node + local_rank-1) % gpus_per_node;
    }
    else if ((gpus_per_node + local_rank - 1)%gpus_per_node  == ring_id)
    {
        send_peer = my_node*gpus_per_node + (local_rank+1) % gpus_per_node;
        recv_peer = (nvshmem_n_pes() + nvshmem_my_pe()-gpus_per_node) % nvshmem_n_pes();
    }
    else
    {
        send_peer = my_node*gpus_per_node + (local_rank+1) % gpus_per_node;
        recv_peer = my_node*gpus_per_node + (gpus_per_node + local_rank-1) % gpus_per_node;
    }

    if(my_node%2 == 1)
        swap_cu(send_peer, recv_peer);

    int stage = 1;
    uint64_t* local_signal = signal + blockIdx.x + blockIdx.y * gridDim.x;
    for (int chunk = 0; chunk < nvshmem_n_pes() - 1; chunk++)
    {
        int send_chunk = (nvshmem_n_pes() + nvshmem_my_pe() - chunk) % nvshmem_n_pes();
        int recv_chunk = (nvshmem_n_pes() + nvshmem_my_pe() - chunk - 1) % nvshmem_n_pes();

        nvshmemx_putmem_signal_nbi_block(destination + off + chunk*chunk_off, buffer + send_chunk*chunk_off + off, block_size,
                local_signal, 1, NVSHMEM_SIGNAL_ADD, send_peer);

        nvshmem_signal_wait_until(local_signal, NVSHMEM_CMP_GE, stage);

        for (int i = threadIdx.x; i < block_size/sizeof(scalar_t); i += blockDim.x)
        {
            float res = float(buffer[recv_chunk*chunk_off + off + i]) + float(destination[off+ chunk*chunk_off + i]);
            buffer[recv_chunk*chunk_off + off + i] = res;
        }
        stage++;
    }

    destination += nvshmem_n_pes() * chunk_off * gridDim.y;

    for (int chunk = 0; chunk < nvshmem_n_pes() - 1; chunk++)
    {
        int send_chunk = (nvshmem_n_pes() + nvshmem_my_pe() - chunk + 1) % nvshmem_n_pes();
        int recv_chunk = (nvshmem_n_pes() + nvshmem_my_pe() - chunk) % nvshmem_n_pes();

        nvshmemx_putmem_signal_nbi_block(destination + off + chunk*chunk_off, buffer + send_chunk*chunk_off + off, block_size,
                local_signal, 1, NVSHMEM_SIGNAL_ADD, send_peer);

        nvshmem_signal_wait_until(local_signal , NVSHMEM_CMP_GE, stage);

        for (int i = threadIdx.x; i < block_size/sizeof(scalar_t); i += blockDim.x)
        {
            buffer[recv_chunk*chunk_off + off + i] = destination[off + chunk*chunk_off + i];
        }
        stage++;
    }
}

void all_reduce_ring(half* buffer, int numel, int packet_size, int block_size, int nnodes, hipStream_t stream) 
{
    // Can we reduce te size of this buffer?
    half *destination = (half *) nvshmem_malloc(2 * numel * sizeof(half));

    nvshmemx_buffer_register(buffer, numel * sizeof(half));
    
    const uint32_t gpus_per_node = nvshmem_n_pes()/nnodes;
    const uint32_t rings = gpus_per_node;
    const uint32_t grid_size_x = std::ceil(numel*sizeof(half) / float(packet_size*block_size*nvshmem_n_pes()*rings));
    dim3 grid_size(grid_size_x, rings, 1);

    uint64_t *signal = (uint64_t *) nvshmem_malloc(grid_size_x * rings * sizeof(uint64_t));
    hipMemset(signal, 0, grid_size_x * 2 * sizeof(uint64_t));
    
    //sync the memset before running kernel
    nvshmemx_barrier_all_on_stream(stream);

    all_reduce_ring_kernel<<<grid_size, block_size, 0, stream>>>(
            destination,
            static_cast<half*>(buffer),
            signal,
            packet_size,
            gpus_per_node
            );

    nvshmemx_barrier_all_on_stream(stream);
    hipStreamSynchronize(stream);

    nvshmemx_buffer_unregister(buffer);
    nvshmem_free(destination);
}
