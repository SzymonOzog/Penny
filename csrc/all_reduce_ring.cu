#include "hip/hip_runtime.h"
#include "device_host_transport/nvshmem_common_transport.h"
#include "host/nvshmem_api.h"
#include "host/nvshmemx_api.h"
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <nvshmem.h>
#include <nvshmemx.h>

template <typename T> __device__ __forceinline__ void swap_cu(T& a, T& b)
{
    T c(a); a=b; b=c;
}

template <typename scalar_t>
__global__ void all_reduce_ring_kernel(scalar_t *destination, scalar_t* buffer, uint64_t* signal, int packet_size, int gpus_per_node) 
{
    const uint64_t base_off = (blockIdx.x * blockDim.x) * packet_size/sizeof(scalar_t);
    const uint64_t block_size = blockDim.x * packet_size;
    const uint64_t chunk_off = (gridDim.x * blockDim.x) * packet_size/sizeof(scalar_t);
    const uint32_t ring_id = blockIdx.y;
    const uint64_t ring_off = ring_id * chunk_off * nvshmem_n_pes();
    const uint64_t off = base_off + ring_off;

    const int pe = nvshmem_my_pe();
    const int n_pes = nvshmem_n_pes();

    const uint32_t local_rank = pe%gpus_per_node;
    const uint32_t my_node = pe/gpus_per_node;

    int send_peer;
    int recv_peer;

    const int M = n_pes / gpus_per_node;
    const int r1 = ((ring_id/2) * 2 + 1);
    const int r_off = (gpus_per_node - r1) % gpus_per_node;
    const int ring_pos = ((( -my_node) % M) * gpus_per_node + ((local_rank - r1) % gpus_per_node) - r_off) % n_pes;

    int send_chunk = ring_pos % n_pes;
    int recv_chunk = (n_pes + ring_pos-1) % n_pes;

    if (local_rank == (ring_id/2)*2)
    {
        send_peer = (n_pes + pe - gpus_per_node+1) % n_pes;
        recv_peer = my_node * gpus_per_node + (local_rank - 1) % gpus_per_node;
    }
    else if (local_rank == (ring_id/2)*2 + 1)
    {
        send_peer = my_node * gpus_per_node + (local_rank + 1) % gpus_per_node;
        recv_peer = (n_pes + pe + gpus_per_node - 1) % n_pes;
    }
    else
    {
        send_peer = my_node*gpus_per_node + (local_rank+1) % gpus_per_node;
        recv_peer = my_node*gpus_per_node + (gpus_per_node + local_rank-1) % gpus_per_node;
    }

    if(ring_id%2 == 1)
    {
        swap_cu(send_chunk, recv_chunk);
        swap_cu(send_peer, recv_peer);
    }

    int stage = 1;
    uint64_t* local_signal = signal + blockIdx.x + blockIdx.y * gridDim.x;
    for (int chunk = 0; chunk < n_pes - 1; chunk++)
    {
        nvshmemx_putmem_signal_nbi_block(destination + off + chunk*chunk_off, buffer + send_chunk*chunk_off + off,
                block_size, local_signal, 1, NVSHMEM_SIGNAL_ADD, send_peer);

        nvshmem_signal_wait_until(local_signal, NVSHMEM_CMP_GE, stage);

        for (int i = threadIdx.x; i < block_size/sizeof(scalar_t); i += blockDim.x)
        {
            float res = float(buffer[recv_chunk*chunk_off + off + i]) + float(destination[off+ chunk*chunk_off + i]);
            buffer[recv_chunk*chunk_off + off + i] = res;
        }
        stage++;
        send_chunk = recv_chunk;
        if(ring_id%2 == 1)
            recv_chunk = (n_pes + recv_chunk + 1)%n_pes;
        else
            recv_chunk = (n_pes + recv_chunk - 1)%n_pes;
    }

    destination += n_pes * chunk_off * gridDim.y;
    for (int chunk = 0; chunk < n_pes - 1; chunk++)
    {
        nvshmemx_putmem_signal_nbi_block(destination + off + chunk*chunk_off, buffer + send_chunk*chunk_off + off,
                block_size, local_signal, 1, NVSHMEM_SIGNAL_ADD, send_peer); 

        nvshmem_signal_wait_until(local_signal , NVSHMEM_CMP_GE, stage);

        for (int i = threadIdx.x; i < block_size/sizeof(scalar_t); i += blockDim.x)
        {
            buffer[recv_chunk*chunk_off + off + i] = destination[off + chunk*chunk_off + i];
        }
        stage++;
        send_chunk = recv_chunk;
        if(ring_id%2 == 1)
            recv_chunk = (n_pes + recv_chunk + 1)%n_pes;
        else
            recv_chunk = (n_pes + recv_chunk - 1)%n_pes;
    }
}

void all_reduce_ring(half* buffer, int numel, int packet_size, int block_size, int nnodes, hipStream_t stream) 
{
    // Can we reduce te size of this buffer?
    half *destination = (half *) nvshmem_malloc(2 * numel * sizeof(half));

    nvshmemx_buffer_register(buffer, numel * sizeof(half));
    
    const uint32_t gpus_per_node = nvshmem_n_pes()/nnodes;
    const uint32_t rings = gpus_per_node;
    const uint32_t grid_size_x = std::ceil(numel*sizeof(half) / float(packet_size*block_size*nvshmem_n_pes()*rings));
    dim3 grid_size(grid_size_x, rings, 1);

    uint64_t *signal = (uint64_t *) nvshmem_malloc(grid_size_x * rings * sizeof(uint64_t));
    hipMemset(signal, 0, grid_size_x * 2 * sizeof(uint64_t));
    
    //sync the memset before running kernel
    nvshmemx_barrier_all_on_stream(stream);

    all_reduce_ring_kernel<<<grid_size, block_size, 0, stream>>>(
            destination,
            static_cast<half*>(buffer),
            signal,
            packet_size,
            gpus_per_node
            );

    nvshmemx_barrier_all_on_stream(stream);
    hipStreamSynchronize(stream);

    nvshmemx_buffer_unregister(buffer);
    nvshmem_free(destination);
}
